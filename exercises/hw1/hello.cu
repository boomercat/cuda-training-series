
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello(){

  printf("Hello from block: %u, thread: %u\n", blockIdx.x,threadIdx.x);
}

int main(){

  hello<<<10,2>>>();
  //host不会等待device执行状态，会直接执行下一条host code。所以为了防止内核打印出其消息前终止，我们需要使用下面的同步函数
  hipDeviceSynchronize();
}

